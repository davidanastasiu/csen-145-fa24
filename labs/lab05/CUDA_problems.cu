#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstdlib>

#define N 1000000  // Array size for vector operations
#define BLOCK_SIZE 256     // Array size for vector operations
#define WIDTH 1024     // Width of matrix/image
#define HEIGHT 1024    // Height of matrix/image 

// Utility function to check for CUDA errors
#define CUDA_CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// 1. Vector Addition Kernel
__global__ void vectorAdd(float* a, float* b, float* c, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    // Your Implementation
}

void vectorAddCPU(float* a, float* b, float* c, int n) {
    for (int i = 0; i < n; ++i) {
        c[i] = a[i] + b[i];
    }
}


__global__ void histogramKernel(int* input, int* histogram, int n, int bins) {
    // Your Implementation
    // Hint: Use AtomicAdd
}

void histogramCPU(int* input, int* histogram, int n, int bins) {
    for (int i = 0; i < bins; ++i) histogram[i] = 0;
    for (int i = 0; i < n; ++i) histogram[input[i]]++;
}

__global__ void nearestNeighborKernel(float* data, float* query, float* result, int n) {
    // Your Implementation
     // Hint: Use AtomicMin
}

void nearestNeighborCPU(float* data, float query, float& result, int n) {
    result = abs(data[0] - query);
    for (int i = 1; i < n; ++i) {
        float diff = abs(data[i] - query);
        if (diff < result) result = diff;
    }
}



__global__ void reverseArray(float* input, float* output, int n) {
    // Your Implementation
}

void reverseArrayCPU(float* input, float* output, int n) {
    for (int i = 0; i < n; ++i) {
        output[n - i - 1] = input[i];
    }
}

__global__ void transposeKernel(float* input, float* output, int width, int height) {
    
    // Your Implementation
}

void transposeCPU(float* input, float* output, int width, int height) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            output[j * height + i] = input[i * width + j];
        }
    }
}

__global__ void convolutionKernel(float* image, float* output, float* filter, int width, int height) {
    // Your Implementation
}

// Convolution (CPU)
void convolutionCPU(float* image, float* output, float* filter, int width, int height) {
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            float sum = 0.0f;
            for (int i = -1; i <= 1; ++i) {
                for (int j = -1; j <= 1; ++j) {
                    int nx = std::min(std::max(x + i, 0), width - 1);
                    int ny = std::min(std::max(y + j, 0), height - 1);
                    sum += image[ny * width + nx] * filter[(i + 1) * 3 + (j + 1)];
                }
            }
            output[y * width + x] = sum;
        }
    }
}


// 7. Bitwise AND Operation
__global__ void bitwiseAnd(int* a, int* b, int* c, int n) {
    // Your Implementation
}

void bitwiseAndCPU(int* a, int* b, int* c, int n) {
    for (int i = 0; i < n; ++i) {
        c[i] = a[i] & b[i];
    }
}
// Utility function to measure time
template <typename Func>
void measureTime(Func func, const char* description) {
    auto start = std::chrono::high_resolution_clock::now();
    func();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;
    std::cout << description << " took " << duration.count() << " ms\n";
}

template <typename T>
void validateResults(T* gpuResult, T* cpuResult, int n, const std::string& operationName) {
    for (int i = 0; i < n; ++i) {
        if (std::abs(gpuResult[i] - cpuResult[i]) > 1e-5) {
            std::cerr << operationName << " failed at index " << i 
                      << ": GPU result = " << gpuResult[i] 
                      << ", CPU result = " << cpuResult[i] << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    std::cout << operationName << " results match!\n";
}

int main() {
    // Allocate and initialize host memory
    float *h_a = new float[N], *h_b = new float[N], *h_cGPU = new float[N], *h_cCPU = new float[N];
    int *h_int_a = new int[N], *h_int_b = new int[N], *h_int_cGPU = new int[N], *h_int_cCPU = new int[N];
    
    for (int i = 0; i < N; ++i) {
        h_a[i] = h_b[i] = static_cast<float>(rand()) / RAND_MAX;
        h_int_a[i] = h_int_b[i] = rand() % 256;
    }

    // Allocate device memory
    int *h_input = new int[N], *h_histogramCPU = new int[256], *h_histogramGPU = new int[256];
    int *d_input, *d_histogram;
    float *d_a, *d_b, *d_c;
    int *d_int_a, *d_int_b, *d_int_c;
    
    CUDA_CHECK(hipMalloc(&d_a, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_c, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_int_a, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_int_b, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_int_c, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_input, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_histogram, 256 * sizeof(int)));
    // -------------------------------------------------------------------------------------------------------------------------//
    // Initialize input for histogram
    for (int i = 0; i < N; ++i) h_input[i] = rand() % 256;
    CUDA_CHECK(hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice));

    // GPU Histogram
    measureTime([&]() {
        CUDA_CHECK(hipMemset(d_histogram, 0, 256 * sizeof(int)));
        histogramKernel<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_input, d_histogram, N, 256);
        CUDA_CHECK(hipDeviceSynchronize());
    }, "GPU Histogram");

    CUDA_CHECK(hipMemcpy(h_histogramGPU, d_histogram, 256 * sizeof(int), hipMemcpyDeviceToHost));

    // CPU Histogram
    measureTime([&]() { histogramCPU(h_input, h_histogramCPU, N, 256); }, "CPU Histogram");
    
    // Validate Histogram Results
    validateResults(h_histogramGPU, h_histogramCPU, 256, "Histogram");

    // Copy data from host to device for vector operations
    CUDA_CHECK(hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_int_a, h_int_a, N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_int_b, h_int_b, N * sizeof(int), hipMemcpyHostToDevice));

    // -------------------------------------------------------------------------------------------------------------------------//

    // Vector Addition
    measureTime([&]() {
        vectorAdd<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
        CUDA_CHECK(hipDeviceSynchronize());
    }, "GPU Vector Addition");

    CUDA_CHECK(hipMemcpy(h_cGPU, d_c, N * sizeof(float), hipMemcpyDeviceToHost));

    measureTime([&]() { vectorAddCPU(h_a, h_b, h_cCPU, N); }, "CPU Vector Addition");
    validateResults(h_cGPU, h_cCPU, N, "Vector Addition");

    // -------------------------------------------------------------------------------------------------------------------------//
    // Bitwise AND
    measureTime([&]() {
        bitwiseAnd<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_int_a, d_int_b, d_int_c, N);
        CUDA_CHECK(hipDeviceSynchronize());
    }, "GPU Bitwise AND");

    CUDA_CHECK(hipMemcpy(h_int_cGPU, d_int_c, N * sizeof(int), hipMemcpyDeviceToHost));

    measureTime([&]() { bitwiseAndCPU(h_int_a, h_int_b, h_int_cCPU, N); }, "CPU Bitwise AND");
    validateResults(h_int_cGPU, h_int_cCPU, N, "Bitwise AND");


    float *h_reverseInput = new float[N];  // Host input array for reverse
    float *h_reverseOutputGPU = new float[N];  // Host GPU result
    float *h_reverseOutputCPU = new float[N];  // Host CPU result

    // Initialize input array with random values
    for (int i = 0; i < N; ++i) {
        h_reverseInput[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory for Array Reverse
    float *d_reverseInput, *d_reverseOutput;
    CUDA_CHECK(hipMalloc(&d_reverseInput, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_reverseOutput, N * sizeof(float)));

    // Copy input data to the device
    CUDA_CHECK(hipMemcpy(d_reverseInput, h_reverseInput, N * sizeof(float), hipMemcpyHostToDevice));

    // Define grid and block dimensions
    // Execute the Array Reverse on GPU
    measureTime([&]() {
        reverseArray<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_reverseInput, d_reverseOutput, N);
        CUDA_CHECK(hipDeviceSynchronize());
    }, "GPU Array Reverse");

    // Copy the result back to the host
    CUDA_CHECK(hipMemcpy(h_reverseOutputGPU, d_reverseOutput, N * sizeof(float), hipMemcpyDeviceToHost));

    // Execute the Array Reverse on CPU
    measureTime([&]() { reverseArrayCPU(h_reverseInput, h_reverseOutputCPU, N); }, "CPU Array Reverse");

    // Validate the Array Reverse Results
    validateResults(h_reverseOutputGPU, h_reverseOutputCPU, N, "Array Reverse");

   // -------------------------------------------------------------------------------------------------------------------------//

    // Nearest Neighbor search
    float query = static_cast<float>(rand()) / RAND_MAX;  // Random query value
    float nearestNeighborResultGPU = std::numeric_limits<float>::max();
    float nearestNeighborResultCPU = std::numeric_limits<float>::max();
    float *d_data, *d_query, *d_result;
    CUDA_CHECK(hipMalloc(&d_data, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_query, sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_result, sizeof(float)));

    // Copy data and query to the device
    CUDA_CHECK(hipMemcpy(d_data, h_a, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_query, &query, sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_result, &nearestNeighborResultGPU, sizeof(float), hipMemcpyHostToDevice));


    // Execute the Nearest Neighbor Search on GPU
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Execute the Nearest Neighbor Search on GPU
    measureTime([&]() {
        nearestNeighborKernel<<<gridSize, blockSize>>>(d_data, d_query, d_result, N);
        CUDA_CHECK(hipDeviceSynchronize());
    }, "GPU Nearest Neighbor Search");
    // Copy the result back to the host
    CUDA_CHECK(hipMemcpy(&nearestNeighborResultGPU, d_result, sizeof(float), hipMemcpyDeviceToHost));

    // Execute the Nearest Neighbor Search on CPU
    measureTime([&]() { nearestNeighborCPU(h_a, query, nearestNeighborResultCPU, N); }, 
                "CPU Nearest Neighbor Search");

    // Validate the Nearest Neighbor Search Results
    if (std::abs(nearestNeighborResultGPU - nearestNeighborResultCPU) > 1e-5) {
        std::cerr << "Nearest Neighbor Search failed: GPU result = " 
                  << nearestNeighborResultGPU << ", CPU result = " 
                  << nearestNeighborResultCPU << std::endl;
    } else {
        std::cout << "Nearest Neighbor Search results match!\n";
    }

    float *h_image = new float[WIDTH * HEIGHT], *h_outputGPU = new float[WIDTH * HEIGHT], *h_outputCPU = new float[WIDTH * HEIGHT];
    float filter[9] = {0, -1, 0, -1, 5, -1, 0, -1, 0};  // Simple sharpening filter

    float *d_image, *d_output, *d_filter;
    CUDA_CHECK(hipMalloc(&d_image, WIDTH * HEIGHT * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, WIDTH * HEIGHT * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_filter, 9 * sizeof(float)));

    // Initialize the image with random values
    for (int i = 0; i < WIDTH * HEIGHT; ++i) h_image[i] = static_cast<float>(rand()) / RAND_MAX;

    // Copy image and filter to device memory
    CUDA_CHECK(hipMemcpy(d_image, h_image, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_filter, filter, 9 * sizeof(float), hipMemcpyHostToDevice));

    // === Convolution Execution and Validation ===
    measureTime([&]() {
        dim3 blockSize(16, 16);
        dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);
        convolutionKernel<<<gridSize, blockSize>>>(d_image, d_output, d_filter, WIDTH, HEIGHT);
        CUDA_CHECK(hipDeviceSynchronize());
    }, "GPU Convolution");

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_outputGPU, d_output, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost));

    // Run convolution on CPU
    measureTime([&]() { convolutionCPU(h_image, h_outputCPU, filter, WIDTH, HEIGHT); }, "CPU Convolution");

    // Validate results
    validateResults(h_outputGPU, h_outputCPU, WIDTH * HEIGHT, "Convolution");

    // -------------------------------------------------------------------------------------------------------------------------//
    // === Matrix Transposition Setup ===
    float *h_matrix = new float[WIDTH * HEIGHT], *h_transposedGPU = new float[WIDTH * HEIGHT], *h_transposedCPU = new float[WIDTH * HEIGHT];

    float *d_matrix, *d_transposed;
    CUDA_CHECK(hipMalloc(&d_matrix, WIDTH * HEIGHT * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_transposed, WIDTH * HEIGHT * sizeof(float)));

    // Initialize the matrix with random values
    for (int i = 0; i < WIDTH * HEIGHT; ++i) h_matrix[i] = static_cast<float>(rand()) / RAND_MAX;

    // Copy matrix to device memory
    CUDA_CHECK(hipMemcpy(d_matrix, h_matrix, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice));

    // === Matrix Transposition Execution and Validation ===
    measureTime([&]() {
        dim3 blockSize(16, 16);
        dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);
        transposeKernel<<<gridSize, blockSize>>>(d_matrix, d_transposed, WIDTH, HEIGHT);
        CUDA_CHECK(hipDeviceSynchronize());
    }, "GPU Matrix Transposition");

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_transposedGPU, d_transposed, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost));

    // Run transposition on CPU
    measureTime([&]() { transposeCPU(h_matrix, h_transposedCPU, WIDTH, HEIGHT); }, "CPU Matrix Transposition");

    // Validate results
    validateResults(h_transposedGPU, h_transposedCPU, WIDTH * HEIGHT, "Matrix Transposition");
    

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    CUDA_CHECK(hipFree(d_int_a));
    CUDA_CHECK(hipFree(d_int_b));
    CUDA_CHECK(hipFree(d_int_c));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_histogram));
    CUDA_CHECK(hipFree(d_image));
    CUDA_CHECK(hipFree(d_transposed));
    CUDA_CHECK(hipFree(d_reverseInput));
    CUDA_CHECK(hipFree(d_reverseOutput));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_filter));
    CUDA_CHECK(hipFree(d_matrix));
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_query));
    CUDA_CHECK(hipFree(d_result));

    delete[] h_a;
    delete[] h_b;
    delete[] h_cGPU;
    delete[] h_cCPU;
    delete[] h_int_a;
    delete[] h_int_b;
    delete[] h_int_cGPU;
    delete[] h_int_cCPU;
    delete[] h_input;
    delete[] h_histogramCPU;
    delete[] h_histogramGPU;
    delete[] h_image;
    delete[] h_outputGPU;
    delete[] h_outputCPU;
    delete[] h_matrix;
    delete[] h_transposedGPU;
    delete[] h_transposedCPU;
    delete[] h_reverseInput;
    delete[] h_reverseOutputGPU;
    delete[] h_reverseOutputCPU;

    return 0;
}
