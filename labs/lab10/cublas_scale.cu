#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define M 1000 
#define N 700

// Function to scale matrix serially
void scaleMatrixSerial(std::vector<float>& matrix, float alpha, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] *= alpha;
    }
}


bool areMatricesEqual(const std::vector<float>& a, const std::vector<float>& b, float tolerance) {
    for (size_t i = 0; i < a.size(); ++i) {
        if (std::fabs(a[i] - b[i]) > tolerance) {
            return false;
        }
    }
    return true;
}

int main() {
    std::vector<float> a(M * N), b(M * N);
    float* devPtrA = nullptr;
    hipblasHandle_t handle;

    // Factor by which to scale the matrix
    float alpha = 10.0f;

   
    srand(42); 

    for (int i = 0; i < M * N; ++i) {
        float randomValue = static_cast<float>(rand()) / RAND_MAX; // Generate a random float between 0 and 1
        a[i] = randomValue;
        b[i] = randomValue; 
    }
        hipEvent_t startCUDA, stopCUDA;
        hipEventCreate(&startCUDA);
        hipEventCreate(&stopCUDA);
        float millisecondsCUDA = 0;
    try {
        hipMalloc((void**)&devPtrA, M * N * sizeof(float));
        hipblasCreate(&handle);
        hipblasSetMatrix(M, N, sizeof(float), a.data(), M, devPtrA, M);
        hipEventRecord(startCUDA);
        // Your implementation
        hipEventRecord(stopCUDA);
        hipEventSynchronize(stopCUDA);
        hipEventElapsedTime(&millisecondsCUDA, startCUDA, stopCUDA);
        std::cout << "GPU Processing time: " << millisecondsCUDA << " ms\n";
        hipblasGetMatrix(M, N, sizeof(float), devPtrA, M, a.data(), M);
        auto startSerial = std::chrono::high_resolution_clock::now();
        scaleMatrixSerial(b, alpha, M, N);
        auto endSerial = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> millisecondsSerial = endSerial - startSerial;
        std::cout << "Serial computation time: " << millisecondsSerial.count() << " ms\n";

        if (areMatricesEqual(a, b, 1e-5)) {
            std::cout << "The matrices are approximately equal." << std::endl;
        } else {
            std::cout << "There is a discrepancy between the matrices." << std::endl;
        }

    } catch (const std::exception& e) {
        std::cerr << "An exception occurred: " << e.what() << std::endl;
        if (devPtrA) hipFree(devPtrA);
        if (handle) hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    
    hipFree(devPtrA);
    hipblasDestroy(handle);
    hipEventDestroy(startCUDA);
    hipEventDestroy(stopCUDA);

    return EXIT_SUCCESS;
}
