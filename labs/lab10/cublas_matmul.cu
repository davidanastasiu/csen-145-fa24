#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstdlib>
#include <cmath>  


bool compareMatrices(float* a, float* b, int n, float tol = 1e-3) {
    for (int i = 0; i < n * n; i++) {
        if (std::fabs(a[i] - b[i]) > tol) {
            return false; 
        }
    }
    return true;  
}

// Error handling macro
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


void matrixMultiplyCPU(float* a, float* b, float* c, int n) {
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            float sum = 0.0;
            for (int k = 0; k < n; k++) {
                sum += a[row * n + k] * b[k * n + col];
            }
            c[row * n + col] = sum;
        }
    }
}

int main() {
    int n = 1000;
    float *a, *b, *c, *c_serial;
    float *d_a, *d_b, *d_c;
    srand(42);
    // Allocate host memory
    a = (float*)malloc(n * n * sizeof(float));
    b = (float*)malloc(n * n * sizeof(float));
    c = (float*)malloc(n * n * sizeof(float));
    c_serial = (float*)malloc(n * n * sizeof(float));

   
    for(int i = 0; i < n * n; i++) {
        a[i] = static_cast<float>(rand()) / RAND_MAX;
        b[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    
    hipMalloc((void**)&d_a, n * n * sizeof(float));
    hipMalloc((void**)&d_b, n * n * sizeof(float));
    hipMalloc((void**)&d_c, n * n * sizeof(float));

    
    hipMemcpy(d_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    
    hipEventRecord(start, 0);
    // Your implementation
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU Elapsed time: " << milliseconds << " ms\n";


    hipMemcpy(c, d_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // CPU version
    auto start_cpu = std::chrono::high_resolution_clock::now();
    matrixMultiplyCPU(a, b, c_serial, n);
    auto stop_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_ms = stop_cpu - start_cpu;
    std::cout << "CPU Elapsed time: " << cpu_ms.count() << " ms\n";

    if (compareMatrices(c, c_serial, n)) {
        std::cout << "The matrices are approximately equal." << std::endl;
    } else {
        std::cout << "There is a discrepancy between the matrices." << std::endl;
    }

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
    free(c_serial);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
